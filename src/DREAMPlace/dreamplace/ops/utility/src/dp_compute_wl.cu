#include "hip/hip_runtime.h"
/* @file: dp_compute_wl.cu
   @date: <date>
*/

#include "utility/src/DetailedPlaceDB.cuh"

DREAMPLACE_BEGIN_NAMESPACE

__global__ void compute_total_hpwl_kernel_edi(DetailedPlaceDB<int> db, const int* xx, const int* yy, int64_t* net_hpwls)
{
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < db.num_nets; i += blockDim.x * gridDim.x)
    {
        net_hpwls[i] = static_cast<int64_t>(db.compute_net_hpwl(i, xx, yy))*HPWLScaleTraits<int>::scale; 
    }
}

int64_t compute_total_hpwl_edi(const DetailedPlaceDB<int>& db, const int* xx, const int* yy, int64_t* net_hpwls)
{
    compute_total_hpwl_kernel_edi<<<CPUCeilDiv(db.num_nets, 512), 512>>>(db, xx, yy, net_hpwls); 

    int64_t* d_out = NULL; 
    // Determine temporary device storage requirements
    void     *d_temp_storage = NULL;
    size_t   temp_storage_bytes = 0;
    hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, net_hpwls, d_out, db.num_nets);
    // Allocate temporary storage
    checkCUDA(hipMalloc(&d_temp_storage, temp_storage_bytes));
    checkCUDA(hipMalloc(&d_out, sizeof(int64_t))); 
    // Run sum-reduction
    hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, net_hpwls, d_out, db.num_nets);
    // copy d_out to hpwl  
    int64_t hpwl = 0; 
    checkCUDA(hipMemcpy(&hpwl, d_out, sizeof(int64_t), hipMemcpyDeviceToHost)); 
    destroyCUDA(d_temp_storage); 
    destroyCUDA(d_out); 

    return hpwl/(db.site_width)/HPWLScaleTraits<int>::scale;
}

DREAMPLACE_END_NAMESPACE
